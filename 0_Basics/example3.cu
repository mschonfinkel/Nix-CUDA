#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addArrays(int* a, int* b, int* c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void){
    const int count = 4;
    int ha[] = {1,2,3,4};
    int hb[] = {10,20,30,40};
    int hc[count];

    int *da, *db, *dc;

    const int gMallocSize = count * sizeof(int);

    hipMalloc(&da, gMallocSize);
    hipMalloc(&db, gMallocSize);
    hipMalloc(&dc, gMallocSize);

    hipMemcpy(da, ha, gMallocSize, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, gMallocSize, hipMemcpyHostToDevice);

    addArrays<<<1, count>>>(da, db, dc);

    // copia os valores de dc de volta para o array do host
    hipMemcpy(hc, dc, gMallocSize, hipMemcpyDeviceToHost);

    // free nos valores alocados pela GPU
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    for(int i = 0; i < count; i++){
        printf("%d ", hc[i]);
    }

    getchar();

    return 0;
}
