#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main(void){
    int count;
    hipGetDeviceCount(&count);

    hipDeviceProp_t prop;

    for(int i=0; i < count; i++){
        hipGetDeviceProperties(&prop, i);
        cout << "Device " << i << ": " << prop.name << endl;
        cout << "Compute Capability: " << prop.major << prop.minor << endl;
        cout << "Max Grid Dimensions: (" <<
            prop.maxGridSize[0] << " x " <<
            prop.maxGridSize[1] << " x " <<
            prop.maxGridSize[2] << " )"  << endl;
        cout << "Max Block Dimensions: (" <<
            prop.maxThreadsDim[0] << " x " <<
            prop.maxThreadsDim[1] << " x " <<
            prop.maxThreadsDim[2] << " )"  << endl;
    }

    getchar();

    return 0;
}
